#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


// template <typename scalar_t>
// __global__ void lltm_cuda_forward_kernel(torch::PackedTensorAccessor32<scalar_t
//             ,2,torch::RestrictPtrTraits> output_gate){
//   int x = blockIdx.x;
//   int y = threadIdx.x;
//   output_gate[x][y] = output_gate[x][y] + 1;
//
//
// }

template <typename scalar_t>
__global__ void kernel(torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> a,
                torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> b,
              torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> out,
            int k){
  int x = blockIdx.x;
  int y = threadIdx.x;
  // assert(a[x][y]!=0);
  float s = 0;
  for(int i=0;i<k;i++){
    s = s + a[b[x][i]][y];
  }
  out[x][y] = s;
}


torch::Tensor gather_reduce_cuda(torch::Tensor a,torch::Tensor b){

  auto shape_a = a.sizes();
  auto shape_b = b.sizes();
  auto out = torch::zeros(shape_a, torch::kFloat).to(torch::kCUDA);

  auto s1 = shape_a[0];
  auto s2 = shape_a[1];
  int s3 = shape_b[0];
  int s4 = shape_b[1];


  // AT_DISPATCH_FLOATING_TYPES(torch::ScalarType::Double, "lltm_forward_cuda", [&]{ lltm_cuda_forward_kernel<scalar_t><<<10, 10 >>> (out.data<scalar_t>());});
  AT_DISPATCH_FLOATING_TYPES(torch::ScalarType::Float, "gather_reduce_cuda", [&]{
     kernel<scalar_t><<<s1, s2 >>> (
       a.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        b.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
        out.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),s4);});


  // AT_DISPATCH_FLOATING_TYPES(torch::ScalarType::Double, "gather_reduce_cuda", ([&] {
  //   lltm_cuda_forward_kernel<scalar_t><<<10, 10 >>>(out.data<scalar_t>);
  // }));
  return out;
}
