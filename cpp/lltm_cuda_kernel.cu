#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


// template <typename scalar_t>
// __global__ void lltm_cuda_forward_kernel(torch::PackedTensorAccessor32<scalar_t
//             ,2,torch::RestrictPtrTraits> output_gate){
//   int x = blockIdx.x;
//   int y = threadIdx.x;
//   output_gate[x][y] = output_gate[x][y] + 1;
//
//
// }

template <typename scalar_t>
__global__ void lltm_cuda_forward_kernel(torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> candidate_cel){
  int x = blockIdx.x;
  int y = threadIdx.x;
  candidate_cel[x][y]=1;
}


torch::Tensor gather_reduce_cuda(torch::Tensor a,torch::Tensor b){

  auto shape_a = a.sizes();
  auto shape_b = b.sizes();
  auto out = torch::zeros(shape_a, torch::kDouble);

  auto s1 = shape_a[0];
  auto s2 = shape_a[1];
  auto s3 = shape_a[0];
  auto s4 = shape_b[1];

  for(int i=0;i<s1;i++){
    for(int j=0;j<s2;j++){
      out.index_put_({i,j},out.index({i,j})+1);
    }
  }

  // AT_DISPATCH_FLOATING_TYPES(torch::ScalarType::Double, "lltm_forward_cuda", [&]{ lltm_cuda_forward_kernel<scalar_t><<<10, 10 >>> (out.data<scalar_t>());});
  AT_DISPATCH_FLOATING_TYPES(torch::ScalarType::Double, "gather_reduce_cuda", [&]{
     lltm_cuda_forward_kernel<scalar_t><<<s1, s2 >>> (out.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());});

  // AT_DISPATCH_FLOATING_TYPES(torch::ScalarType::Double, "gather_reduce_cuda", ([&] {
  //   lltm_cuda_forward_kernel<scalar_t><<<10, 10 >>>(out.data<scalar_t>);
  // }));
  // return out;
}
